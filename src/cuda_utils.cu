#include "hip/hip_runtime.h"
#include <opencv2/opencv.hpp>
#include "cuda_utils.h"

__global__ void grayscaleKernel(unsigned char* d_input, unsigned char* d_output, int width, int height, int channels) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x < width && y < height) {
        int idx = (y * width + x) * channels;
        unsigned char r = d_input[idx];
        unsigned char g = d_input[idx + 1];
        unsigned char b = d_input[idx + 2];
        d_output[y * width + x] = 0.299f * r + 0.587f * g + 0.114f * b;
    }
}

void cudaGrayscale(const cv::Mat& input, cv::Mat& output) {
    int width = input.cols;
    int height = input.rows;
    int channels = input.channels();

    output.create(height, width, CV_8UC1);

    size_t inputSize = width * height * channels;
    size_t outputSize = width * height;

    unsigned char *d_input, *d_output;
    hipMalloc(&d_input, inputSize);
    hipMalloc(&d_output, outputSize);

    hipMemcpy(d_input, input.data, inputSize, hipMemcpyHostToDevice);

    dim3 blockSize(16, 16);
    dim3 gridSize((width + blockSize.x - 1) / blockSize.x, (height + blockSize.y - 1) / blockSize.y);

    grayscaleKernel<<<gridSize, blockSize>>>(d_input, d_output, width, height, channels);

    hipMemcpy(output.data, d_output, outputSize, hipMemcpyDeviceToHost);

    hipFree(d_input);
    hipFree(d_output);
}